#include <stdio.h>
#include "VectorTypesUtil.h"

#include <thread>
#include "Util.h"
#include "Camera.h"
#include "OpenGL/GlWindow.h"
#include "OpenGL/GlTexture2d.h"
#include "CUDA/CudaCubeMap.h"
#include "PathTracing/ProgressivePathTracing.h"

//#include <GLFW/glfw3.h>
#include "../packages/glm.0.9.7.1/build/native/include/glm/detail/type_vec3.hpp"
#include <algorithm>

//#include <Windows.h>

#include "hip/hip_runtime.h"
#include ""
#include "surface_functions.hpp"
#include "surface_indirect_functions.hpp"
#include "hip/hip_complex.h"
#include <hiprand/hiprand_kernel.h>
#include <cfloat>
#include "CUDA/CudaSurfaceObject.h"
#include "PathTracing/ProgressiveTracer.h"

int main(int argc, char** argv)
{

   // Choose which GPU to run on, change this on a multi-GPU system.
   auto cudaStatus = hipSetDevice( 0 );
   if ( cudaStatus != hipSuccess )
   {
      fprintf( stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" );
      return;
   }

   ProgressiveTracer tracer( 512, 512 );
   tracer.Run();
   //InteractiveTracer();

   // hipDeviceReset must be called before exiting in order for profiling and
   // tracing tools such as Nsight and Visual Profiler to show complete traces.
   cudaStatus = hipDeviceReset();
   if ( cudaStatus != hipSuccess )
   {
      fprintf( stderr, "hipDeviceReset failed!" );
   }

   return 0;
}
